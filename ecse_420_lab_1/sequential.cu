
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define AND 0
#define OR 1
#define NAND 2
#define NOR 3
#define XOR 4
#define XNOR 5

int gate(int a, int b, int type) {
  switch (type) {
  case AND: return a & b;
  case OR: return a | b;
  case NAND: return !(a & b);
  case NOR: return !(a | b);
  case XOR: return a ^ b;
  case XNOR: return !(a ^ b);
  default: return -1;
  }
}

void simulate_gates(int* output, const int* input, int len) {
  for (int i = 0; i < len; i++) {
    output[i] = gate(input[i * 3], input[i * 3 + 1], input[i * 3 + 2]);
  }
}

int main(int argc, char* argv[]) {
  if (argc != 4) {
    printf("Invalid number of program arguments");
    exit(EXIT_FAILURE);
  }

  char* input_path = argv[1];
  int input_len = atoi(argv[2]);
  char* output_path = argv[3];
  if (input_len <= 0) {
    printf("Invalid input length");
    exit(EXIT_FAILURE);
  }


  int* inputs = (int*)malloc((long long)3 * input_len * sizeof(int));
  int* outputs = (int*)malloc(input_len * sizeof(int));
  if (inputs == NULL || outputs == NULL) {
    printf("Could not allocate memory for gate simulation");
    exit(EXIT_FAILURE);
  }

  FILE* input = fopen(input_path, "r");
  FILE* output = fopen(output_path, "w");
  if (input == NULL || output == NULL) {
    printf("Could not open files for reading or writing");
    exit(EXIT_FAILURE);
  }

  for (int i = 0; i < input_len; i++) {
    char buf[7];
    fgets(buf, 7, input);
    inputs[i * 3] = (int)(buf[0] - '0');
    inputs[i * 3 + 1] = (int)(buf[2] - '0');
    inputs[i * 3 + 2] = (int)(buf[4] - '0');
  }

  simulate_gates(outputs, inputs, input_len);

  for (int i = 0; i < input_len; i++) {
    char line[] = {outputs[i] + '0', '\n', '\0'};
    fputs(line, output);
  }

  free(inputs);
  free(outputs);

  fclose(input);
  fclose(output);
  
  return 0;
}
