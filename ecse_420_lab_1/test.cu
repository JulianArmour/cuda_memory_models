#include "hip/hip_runtime.h"

#include <stdio.h>

__global__ void hi() {
  printf("Hello from the gpu!\n");
}

void sayhi() {
  hi <<<1, 1>>> ();
}