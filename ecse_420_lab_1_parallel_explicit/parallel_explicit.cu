#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"


#define AND 0
#define OR 1
#define NAND 2
#define NOR 3
#define XOR 4
#define XNOR 5

__device__ int gate_gpu(int a, int b, int type) {
  switch (type) {
  case AND: return a & b;
  case OR: return a | b;
  case NAND: return !(a & b);
  case NOR: return !(a | b);
  case XOR: return a ^ b;
  case XNOR: return !(a ^ b);
  default: return -1;
  }
}

__global__ void simulate_gates_gpu(int* output, const int* input, int len) {
  int i = blockIdx.x;
  if (i >= len)
    return;
  output[i] = gate_gpu(input[i * 3], input[i * 3 + 1], input[i * 3 + 2]);
}

void simulate_gates(int* output, const int* input, int len) {
  int * d_input, * d_output;
  hipMalloc(&d_input, (long long)3 * len * sizeof(int));
  hipMalloc(&d_output,  len * sizeof(int));

  // record time taken to transfer data to device and run the kernel.
  //start timer
  float memsettime;
  hipEvent_t start, stop;
  hipEventCreate(&start); hipEventCreate(&stop);
  hipEventRecord(start, 0);

  hipMemcpy(d_input, input, (long long)3 * len * sizeof(int), hipMemcpyHostToDevice);
  simulate_gates_gpu<<<len,1>>>(d_output, d_input, len);

  //stop timer
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&memsettime, start, stop);
  printf("Kernel execution time: %f\n", memsettime);
  hipEventDestroy(start); hipEventDestroy(stop);

  hipMemcpy(output, d_output, len * sizeof(int), hipMemcpyDeviceToHost);
  hipFree(d_input);
  hipFree(d_output);
}

int main(int argc, char* argv[]) {
  if (argc != 4) {
    printf("Invalid number of program arguments");
    exit(EXIT_FAILURE);
  }

  char* input_path = argv[1];
  int input_len = atoi(argv[2]);
  char* output_path = argv[3];
  if (input_len <= 0) {
    printf("Invalid input length");
    exit(EXIT_FAILURE);
  }


  int* inputs = (int*)malloc((long long)3 * input_len * sizeof(int));
  int* outputs = (int*)malloc(input_len * sizeof(int));
  if (inputs == NULL || outputs == NULL) {
    printf("Could not allocate memory for gate simulation");
    exit(EXIT_FAILURE);
  }

  FILE* input = fopen(input_path, "r");
  FILE* output = fopen(output_path, "w");
  if (input == NULL || output == NULL) {
    printf("Could not open files for reading or writing");
    exit(EXIT_FAILURE);
  }

  for (int i = 0; i < input_len; i++) {
    char buf[7];
    fgets(buf, 7, input);
    inputs[i * 3] = (int)(buf[0] - '0');
    inputs[i * 3 + 1] = (int)(buf[2] - '0');
    inputs[i * 3 + 2] = (int)(buf[4] - '0');
  }

  simulate_gates(outputs, inputs, input_len);

  for (int i = 0; i < input_len; i++) {
    char line[] = {outputs[i] + '0', '\n', '\0'};
    fputs(line, output);
  }

  free(inputs);
  free(outputs);

  fclose(input);
  fclose(output);
  
  return 0;
}
